#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    cuda_hello<<<1,1>>>();
    printf("Hello World from CPU!\n");
    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}

/*
compile:    nvcc -arch compute_50 hello.cu
run:        ./a.out 
*/