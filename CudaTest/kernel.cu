#include "hip/hip_runtime.h"

#include <stdio.h>

const int allocating_size = 1 * 1024 * 1024 * 1024;

void *cpu_p;
void *gpu_p;

void cpu_alloc()
{
    cpu_p = malloc(allocating_size);
}

void gpu_alloc()
{
    hipError_t sonuc = hipMalloc(&gpu_p, allocating_size);
    printf("%d\n",sonuc);
    // assert(sonuc == cudaSuccess); //! 
}

int main()
{
    cpu_alloc();

    gpu_alloc();

    return 0;
}

/*
compile:    nvcc -arch compute_50 kernel.cu
run:        ./a.out 
*/
