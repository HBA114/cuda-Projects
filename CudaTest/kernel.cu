#include "hip/hip_runtime.h"

#include <stdio.h>

const int alan_boyut = 1 * 1024 * 1024 * 1024;

void *cpu_p;
void *gpu_p;

void cpu_alloc()
{
    cpu_p = malloc(alan_boyut);
}

void gpu_alloc()
{
    hipError_t sonuc = hipMalloc(&gpu_p, alan_boyut);
    printf("%d\n",sonuc);
    // assert(sonuc == cudaSuccess); //! 
}

int main()
{
    cpu_alloc();

    gpu_alloc();

    return 0;
}
/*

compile:    nvcc -arch compute_50 kernel.cu
run:        ./a.out 


*/
