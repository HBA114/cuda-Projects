#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

const int number = 1024;
const int allocating_size = number * sizeof(int);

void *cpu_p;
void *gpu_p;

void cpu_alloc()
{
    cpu_p = malloc(allocating_size);
}

void cpu_set_numbers()
{
    int *cpu_int32 = (int *)cpu_p;
    for (int i = 0; i < number; i++)
        cpu_int32[i] = (i + 1) * 2;
}

void cpu_free()
{
    free(cpu_p);
}

void cpu_memory_to_gpu_memory()
{
    hipError_t result = hipMemcpy(gpu_p, cpu_p, allocating_size, hipMemcpyHostToDevice);
    assert(result == hipSuccess);
}

void gpu_memory_to_cpu_memory()
{
    hipError_t result = hipMemcpy(cpu_p, gpu_p, allocating_size, hipMemcpyDeviceToHost);
    assert(result == hipSuccess);
}

void gpu_alloc()
{
    hipError_t result = hipMalloc(&gpu_p, allocating_size);
    assert(result == hipSuccess);
}

void gpu_free()
{
    hipError_t result = hipFree(gpu_p);
    assert(result == hipSuccess);
}

__global__ void gpu_add(int *gpu_numbers)
{
    int id = threadIdx.x;

    gpu_numbers[id] = gpu_numbers[id] + 100;
}

int main()
{
    cpu_alloc();
    cpu_set_numbers();

    int *cpu_int32 = (int *)cpu_p;
    for (int i = 0; i < number; i++)
        printf("%d \n", cpu_int32[i]);

    printf("\n ------------------------------ \n\n");
    gpu_alloc();
    cpu_memory_to_gpu_memory();

    // execute
    //! if nnumber is greater than 1024,
    //! gpu_add will not executed as expected
    gpu_add<<<1, number>>>((int *)gpu_p);

    hipError_t result = hipDeviceSynchronize();
    assert(result == hipSuccess);

    gpu_memory_to_cpu_memory();

    cpu_int32 = (int *)cpu_p;
    for (int i = 0; i < number; i++)
        printf("%d \n", cpu_int32[i]);

    gpu_free();
    cpu_free();

    printf("Completed.\n");

    return 0;
}
/*
compile: nvcc -arch compute_50 kernerl.cu
*/
